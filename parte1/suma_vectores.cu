#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                         \
do {                                                                             \
    hipError_t _e = (call);                                                     \
    if (_e != hipSuccess) {                                                     \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,            \
                hipGetErrorString(_e));                                         \
        exit(EXIT_FAILURE);                                                      \
    }                                                                            \
} while (0)

__global__ void vecAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main(int argc, char** argv) {
    // N por defecto ~1M; puedes pasar N, blockSize por línea de comandos.
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 20);
    int blockSize = (argc > 2) ? atoi(argv[2]) : 256;
    size_t bytes = (size_t)N * sizeof(float);

    // Declarar e inicializar en host
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    if (!h_A || !h_B || !h_C) {
        fprintf(stderr, "Fallo al reservar memoria en host\n");
        return EXIT_FAILURE;
    }
    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = (float)i * 0.001f;
    }

    // Reservar en device
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_C, bytes));

    // Copiar host -> device
    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    // Configurar grid/block
    int gridSize = (N + blockSize - 1) / blockSize;

    // medir tiempo del kernel con eventos
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    // Lanzar kernel en GPU
    vecAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));

    // Copiar device -> host
    CUDA_CHECK(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    // Validación rápida
    bool ok = true;
    for (int i = 0; i < 10 && i < N; ++i) {
        float expected = h_A[i] + h_B[i];
        if (fabs(h_C[i] - expected) > 1e-5f) { ok = false; break; }
    }

    printf("N=%d | grid=%d block=%d | tiempo kernel = %.3f ms | resultado %s\n",
           N, gridSize, blockSize, ms, ok ? "OK" : "MAL");

    // Imprime algunos resultados
    for (int i = 0; i < 5 && i < N; ++i) {
        printf("C[%d] = %.6f\n", i, h_C[i]);
    }

    // Limpieza
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(h_A); free(h_B); free(h_C);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    return ok ? EXIT_SUCCESS : EXIT_FAILURE;
}
